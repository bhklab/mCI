#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>



static const char *curandGetErrorString(hiprandStatus_t error)
{
	switch (error)
	{
		case HIPRAND_STATUS_SUCCESS:
			return "CURAND_STATUS_SUCCESS";

		case HIPRAND_STATUS_VERSION_MISMATCH:
			return "CURAND_STATUS_VERSION_MISMATCH";

		case HIPRAND_STATUS_NOT_INITIALIZED:
			return "CURAND_STATUS_NOT_INITIALIZED";

		case HIPRAND_STATUS_ALLOCATION_FAILED:
			return "CURAND_STATUS_ALLOCATION_FAILED";

		case HIPRAND_STATUS_TYPE_ERROR:
			return "CURAND_STATUS_TYPE_ERROR";

		case HIPRAND_STATUS_OUT_OF_RANGE:
			return "CURAND_STATUS_OUT_OF_RANGE";

		case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
			return "CURAND_STATUS_LENGTH_NOT_MULTIPLE";

		case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
			return "CURAND_STATUS_DOUBLE_PRECISION_REQUIRED";

		case HIPRAND_STATUS_LAUNCH_FAILURE:
			return "CURAND_STATUS_LAUNCH_FAILURE";

		case HIPRAND_STATUS_PREEXISTING_FAILURE:
			return "CURAND_STATUS_PREEXISTING_FAILURE";

		case HIPRAND_STATUS_INITIALIZATION_FAILED:
			return "CURAND_STATUS_INITIALIZATION_FAILED";

		case HIPRAND_STATUS_ARCH_MISMATCH:
			return "CURAND_STATUS_ARCH_MISMATCH";

		case HIPRAND_STATUS_INTERNAL_ERROR:
			return "CURAND_STATUS_INTERNAL_ERROR";
	}

	return "<unknown>";
}


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	   if (code != hipSuccess) 
		      {
			            fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
				          if (abort) exit(code);
					     }
}

inline void gpuAssertRand(hiprandStatus_t code, const char *file, int line, bool abort=true)
{
	if (code != HIPRAND_STATUS_SUCCESS)
		{
		        fprintf(stderr,"GPUassert: %s %s %d\n", curandGetErrorString(code), file, line);
			if (abort) exit(code);
	       	}
}
