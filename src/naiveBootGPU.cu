#include "hip/hip_runtime.h"
/*
// Fast permutations for rCI using a naive matrix based approach.
*/


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <R.h>
#include <Rinternals.h>
// #include "xoroshiro128+.h"

 #define min(a,b) \
   ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
     _a < _b ? _a : _b; })

static const char *curandGetErrorString(hiprandStatus_t error)
{
	switch (error)
	{
		case HIPRAND_STATUS_SUCCESS:
			return "HIPRAND_STATUS_SUCCESS";

		case HIPRAND_STATUS_VERSION_MISMATCH:
			return "HIPRAND_STATUS_VERSION_MISMATCH";

		case HIPRAND_STATUS_NOT_INITIALIZED:
			return "HIPRAND_STATUS_NOT_INITIALIZED";

		case HIPRAND_STATUS_ALLOCATION_FAILED:
			return "HIPRAND_STATUS_ALLOCATION_FAILED";

		case HIPRAND_STATUS_TYPE_ERROR:
			return "HIPRAND_STATUS_TYPE_ERROR";

		case HIPRAND_STATUS_OUT_OF_RANGE:
			return "HIPRAND_STATUS_OUT_OF_RANGE";

		case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
			return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";

		case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
			return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";

		case HIPRAND_STATUS_LAUNCH_FAILURE:
			return "HIPRAND_STATUS_LAUNCH_FAILURE";

		case HIPRAND_STATUS_PREEXISTING_FAILURE:
			return "HIPRAND_STATUS_PREEXISTING_FAILURE";

		case HIPRAND_STATUS_INITIALIZATION_FAILED:
			return "HIPRAND_STATUS_INITIALIZATION_FAILED";

		case HIPRAND_STATUS_ARCH_MISMATCH:
			return "HIPRAND_STATUS_ARCH_MISMATCH";

		case HIPRAND_STATUS_INTERNAL_ERROR:
			return "HIPRAND_STATUS_INTERNAL_ERROR";
	}

	return "<unknown>";
}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	   if (code != hipSuccess) 
		      {
			            fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
				          if (abort) exit(code);
					     }
}

#define gpuErrchkRand(ans) { gpuAssertRand((ans), __FILE__, __LINE__); }
inline void gpuAssertRand(hiprandStatus_t code, const char *file, int line, bool abort=true)
{
	if (code != HIPRAND_STATUS_SUCCESS)
		{
		        fprintf(stderr,"GPUassert: %s %s %d\n", curandGetErrorString(code), file, line);
			if (abort) exit(code);
	       	}
}

// #define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
//     printf("Error at %s:%d\n",__FILE__,__LINE__);\
//     return EXIT_FAILURE;}} while(0)


const int numThreads = 64;


// Code to create indicies properly from the uniform random numbers. 
__global__
void truncate_to_index(double *randomDoubles, uint64_t *randomInt, uint64_t N, uint64_t maxI){
   
  uint64_t i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i >= maxI){
   return;
  }
  randomInt[i] = (uint64_t)ceil(randomDoubles[i] * N) - 1;

}



__global__
void runBootOnDevice(double *rcimat, double *outVec, uint64_t *permVector, uint64_t N, uint64_t R){
  
  uint64_t i = blockIdx.x*blockDim.x + threadIdx.x;
  
  if(i >= R){
    return;
  }

  uint64_t *permIdx;

  double currCI;
  double curVal;
  double RS_numerator, RS_denominator;
  RS_numerator = 0;
  RS_denominator = 0;
  permIdx = permVector + i*N;
  
  for(uint64_t j = 0; j < N; j++){

      for(uint64_t k = 0; k < N; k++){
        curVal = rcimat[permIdx[j]*N + permIdx[k]];

        RS_numerator += (curVal * (double)(curVal > 0));
        RS_denominator += (double)(curVal != 0) * 2;
      }

    }

    currCI = (RS_numerator)/(RS_denominator);
    
    outVec[i] = currCI;

}


void bootOnCuda(double *rcimat, double *outVec, uint64_t R, uint64_t N, int xties, int yties, uint64_t *state){


  double *devrcimat, *devOutVec, *devRandomNumbers;

  hiprandGenerator_t gen;
  size_t free_mem, total_mem;
  size_t mem_needed_per_R;

  uint64_t *permVector;

  uint64_t RperLoop, Roffset, loopI;

  gpuErrchkRand(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

  hipDeviceSynchronize();
  gpuErrchkRand(hiprandSetPseudoRandomGeneratorSeed(gen, *state));
  hipDeviceSynchronize();

  // Need to calculate here to make sure we don't run out of memory while using the GPU. 
  // Will do the computation in batches over R. 

  gpuErrchk(hipMalloc(&devrcimat, N*N*sizeof(double))); 
  gpuErrchk(hipMemcpy(devrcimat, rcimat, N*N*sizeof(double), hipMemcpyHostToDevice));


  gpuErrchk(hipMemGetInfo(&free_mem, &total_mem));



  mem_needed_per_R = (2*(N)+1)*sizeof(double);

  free_mem = free_mem - 50*sizeof(double); // keeping some extra buffer space of 50 doubles for variables allocated in kernels

  RperLoop = min(free_mem / mem_needed_per_R, R);

  printf("R per loop: %lld", RperLoop);

  for(loopI = 0; loopI < (R/RperLoop)+1; loopI++){

    Roffset = loopI * RperLoop;

    if((RperLoop + Roffset) > R){
      RperLoop = R - Roffset;
    }
    if(RperLoop == 0){
      break
    }

    gpuErrchk(hipMalloc(&devOutVec, RperLoop*sizeof(double)));

    gpuErrchk(hipMalloc(&devRandomNumbers, RperLoop*N*sizeof(double)));
    gpuErrchk(hipMalloc(&permVector, RperLoop*N*sizeof(uint64_t)));






    gpuErrchkRand(hiprandGenerateUniformDouble(gen, devRandomNumbers, RperLoop*N));
    hipDeviceSynchronize();
  // Creating permutation indicies from uniform doubles
    truncate_to_index<<<(RperLoop*N+(numThreads-1))/numThreads, numThreads>>>(devRandomNumbers, permVector, N, RperLoop*N);
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
            // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

    // Running one bootstrap instance per thread.  
    runBootOnDevice<<<(RperLoop+(numThreads-1))/numThreads, numThreads>>>(devrcimat, devOutVec, permVector, N, RperLoop);
    hipDeviceSynchronize();

    error = hipGetLastError();
    if(error != hipSuccess) {
            // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
    //Copying back results
    gpuErrchk(hipMemcpy(outVec+Roffset, devOutVec, RperLoop*sizeof(double), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    
    gpuErrchk(hipFree(permVector));
    gpuErrchk(hipFree(devRandomNumbers));
    gpuErrchk(hipFree(devOutVec));
  }

  

  // Freeing Memory
  
  gpuErrchk(hipFree(devrcimat));
  hiprandDestroyGenerator(gen);

}

extern "C"
SEXP bootCUDA(SEXP prcimat,
             // SEXP pobsCI,
             SEXP pR,
             // SEXP pB,
             SEXP pn,
             SEXP pxties,
             SEXP pyties,
             SEXP pseed){
  
  double Ndouble = *REAL(pn);
  
  // int discard_x_ties = *INTEGER(pdiscard_x_ties);
  // int discard_y_ties = *INTEGER(pdiscard_y_ties);
  
  // double obsCI = *REAL(pobsCI);

  double Rdouble = *REAL(pR);  
  // double Bdouble = *REAL(pB);

  uint64_t N = (uint64_t) Ndouble;
  uint64_t R = (uint64_t) Rdouble;
  // uint64_t B = (uint64_t) Bdouble;

  int xties = *INTEGER(pxties);
  int yties = *INTEGER(pyties);
  
  SEXP pout = PROTECT(allocVector(REALSXP,R));
  
  // double *out = REAL(pout);
  
  double *seed = REAL(pseed);
  uint64_t *state = (uint64_t*) seed;

  // double *rcimat2 = malloc(N * N * sizeof(double));
  bootOnCuda(REAL(prcimat), REAL(pout), R, N, xties, yties, state);
  // printf("%f\n", out[0]);
  // rciBootWithCopy(REAL(prcimat), rcimat2, REAL(pout), R, N, xties, yties, state);


  // free(rcimat2);
  UNPROTECT(1);
  
  return pout;
  
}


